
#include <hip/hip_runtime.h>
__global__ void add_kernel(float *c, 
    const float *a, 
    const float *b, 
    int n) {
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
            c[i] = a[i] + b[i];
        }
    }

void add_op(float *c, 
    const float *a, 
    const float *b, 
    int n) {
        dim3 grid(n+1023/1024);
        dim3 block(1024);
        add_kernel<<<grid, block>>>(c, a, b, n);
    }
