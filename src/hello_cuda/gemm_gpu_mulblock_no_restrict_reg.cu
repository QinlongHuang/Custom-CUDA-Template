#include "hip/hip_runtime.h"
#include "gemm.h"

__global__
void gemm_gpu_mulblock_no_restrict_reg_kernel(
	int* C,		// [n, m]
	const int* A,	// [n, k]
	const int* B,	// [k, m]
	const int n,
	const int m,
	const int k
) {
	const int i = threadIdx.x;
    const int j = blockIdx.x;
    int res = 0;
    for (int l = 0; l < k; l++) {
        res += A[i*k + l] * B[l*m + j];
    }
    C[i*m + j] = res;
}

void gemm_gpu_mulblock_no_restrict_reg(
	int* C,		// [n, m]
	const int* A,	// [n, k]
	const int* B,	// [k, m]
	const int n,
	const int m,
	const int k
) {
	gemm_gpu_mulblock_no_restrict_reg_kernel<<<m, n>>>(C, A, B, n, m, k);
}